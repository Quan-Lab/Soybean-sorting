#include "hip/hip_runtime.h"
#include "MWBatchNormalizationLayer.hpp"
#include "MWBatchNormalizationLayerImpl.hpp"
#include <stdio.h>
#include <cassert> 
 __global__ __launch_bounds__(1024) void computeBNParams(float* 
pxmnUEWGnfCxJNuDkXAo,  float* hYTzvgWajqchLzrmxjqn,  float* vcOGADqMrTrPPcuYvrHS,  
float* vlzcDcTSrYXZiamsmNlx,  float RgALmBtPIZWDevjZBUHy, float* 
qLXeoFROCbISdsnwpYgl, float* sXWXkiDEKpurgeCqZLDL, float* 
niGnnRufksTFnsUUxnCj, int numVals) { long unsigned int SmibqCQPbtzycGEpwhpN = 
blockIdx.x*blockDim.x + threadIdx.x; for(; SmibqCQPbtzycGEpwhpN < numVals; 
SmibqCQPbtzycGEpwhpN+= blockDim.x*gridDim.x) { 
qLXeoFROCbISdsnwpYgl[SmibqCQPbtzycGEpwhpN] = 
pxmnUEWGnfCxJNuDkXAo[SmibqCQPbtzycGEpwhpN]/sqrt(vlzcDcTSrYXZiamsmNlx[SmibqCQPbtzycGEpwhpN] 
+ RgALmBtPIZWDevjZBUHy); sXWXkiDEKpurgeCqZLDL[SmibqCQPbtzycGEpwhpN] = 
hYTzvgWajqchLzrmxjqn[SmibqCQPbtzycGEpwhpN] - 
(vcOGADqMrTrPPcuYvrHS[SmibqCQPbtzycGEpwhpN]*qLXeoFROCbISdsnwpYgl[SmibqCQPbtzycGEpwhpN]); 
niGnnRufksTFnsUUxnCj[SmibqCQPbtzycGEpwhpN] = 1.f; } } 
MWBatchNormalizationLayerImpl::MWBatchNormalizationLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, double const KnbXZOZzgyMJTNGorRue, const 
char* LvFwgsCgepJxzddrrZNH, const char* LzeBIFvfQmAPApScLUyy, const char* 
MXVlAOBGDMdzlrCMvAgl, const char* MmeSJXZBVyUbgMuuFbuc, int 
, int ) : MWCNNLayerImpl(layer, ntwk_impl)  , pxmnUEWGnfCxJNuDkXAo(NULL) , 
hYTzvgWajqchLzrmxjqn(NULL) , vcOGADqMrTrPPcuYvrHS(NULL) , 
vlzcDcTSrYXZiamsmNlx(NULL) , qLXeoFROCbISdsnwpYgl(NULL) , 
sXWXkiDEKpurgeCqZLDL(NULL) , niGnnRufksTFnsUUxnCj(NULL) { 
MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* ipTensor = 
BNLayer->getInputTensor(); MWTensor* opTensor = BNLayer->getOutputTensor(); 
RgALmBtPIZWDevjZBUHy = KnbXZOZzgyMJTNGorRue; const int vVyVzWKKaCvGClCSagOb = 
2048; QJJBjzDRkBQsCLkHaADa = (ipTensor->getChannels() <= 
vVyVzWKKaCvGClCSagOb)?false:true; const size_t eczxzisTMvVejXfupdkv = 
sizeof(float)*ipTensor->getChannels(); qLXeoFROCbISdsnwpYgl = 
(float*)malloc(eczxzisTMvVejXfupdkv); sXWXkiDEKpurgeCqZLDL = 
(float*)malloc(eczxzisTMvVejXfupdkv); niGnnRufksTFnsUUxnCj = 
(float*)malloc(eczxzisTMvVejXfupdkv); if (QJJBjzDRkBQsCLkHaADa) { 
hipMalloc(&pxmnUEWGnfCxJNuDkXAo, eczxzisTMvVejXfupdkv); 
hipMalloc(&hYTzvgWajqchLzrmxjqn, eczxzisTMvVejXfupdkv); 
hipMalloc(&vcOGADqMrTrPPcuYvrHS, eczxzisTMvVejXfupdkv); 
hipMalloc(&vlzcDcTSrYXZiamsmNlx, eczxzisTMvVejXfupdkv); 
loadScale(LzeBIFvfQmAPApScLUyy); loadOffset(LvFwgsCgepJxzddrrZNH); 
loadTrainedMean(MXVlAOBGDMdzlrCMvAgl); 
loadTrainedVariance(MmeSJXZBVyUbgMuuFbuc); float* 
qMpgAuYpEDGDohMcPvRY = NULL; float* siiUXAavgXoUUOVaXdoz = NULL; float* 
nrUaguRlWYtZgNEGzlhH = NULL; hipMalloc(&qMpgAuYpEDGDohMcPvRY, 
eczxzisTMvVejXfupdkv); hipMalloc(&siiUXAavgXoUUOVaXdoz, 
eczxzisTMvVejXfupdkv); hipMalloc(&nrUaguRlWYtZgNEGzlhH, 
eczxzisTMvVejXfupdkv); int fPIxBBGHjPkvmoaWByBr = ipTensor->getChannels(); int 
uTUuLVVebDakbPjXOQwp = 
std::floor(static_cast<float>(fPIxBBGHjPkvmoaWByBr)/static_cast<float>(32)) * 32; 
int uqHugYAAqkSnCCYonqCt = (uTUuLVVebDakbPjXOQwp < 1024) ? uTUuLVVebDakbPjXOQwp : 
1024; int OJTEGflbxqozjWWEaUJd = (fPIxBBGHjPkvmoaWByBr + 
uqHugYAAqkSnCCYonqCt - 1)/uqHugYAAqkSnCCYonqCt; 
computeBNParams<<<OJTEGflbxqozjWWEaUJd,uqHugYAAqkSnCCYonqCt>>>(pxmnUEWGnfCxJNuDkXAo, 
 hYTzvgWajqchLzrmxjqn,  vcOGADqMrTrPPcuYvrHS,  vlzcDcTSrYXZiamsmNlx, 
RgALmBtPIZWDevjZBUHy, qMpgAuYpEDGDohMcPvRY, siiUXAavgXoUUOVaXdoz, 
nrUaguRlWYtZgNEGzlhH, fPIxBBGHjPkvmoaWByBr); 
hipMemcpy(qLXeoFROCbISdsnwpYgl, qMpgAuYpEDGDohMcPvRY, 
eczxzisTMvVejXfupdkv, hipMemcpyDeviceToHost); 
hipMemcpy(sXWXkiDEKpurgeCqZLDL, siiUXAavgXoUUOVaXdoz, 
eczxzisTMvVejXfupdkv, hipMemcpyDeviceToHost); 
hipMemcpy(niGnnRufksTFnsUUxnCj, nrUaguRlWYtZgNEGzlhH, 
eczxzisTMvVejXfupdkv, hipMemcpyDeviceToHost); 
hipFree(qMpgAuYpEDGDohMcPvRY); hipFree(siiUXAavgXoUUOVaXdoz); 
hipFree(nrUaguRlWYtZgNEGzlhH); } else { const size_t eczxzisTMvVejXfupdkv = 
sizeof(float)*ipTensor->getChannels(); pxmnUEWGnfCxJNuDkXAo = 
(float*)malloc(eczxzisTMvVejXfupdkv); hYTzvgWajqchLzrmxjqn = 
(float*)malloc(eczxzisTMvVejXfupdkv); vcOGADqMrTrPPcuYvrHS = 
(float*)malloc(eczxzisTMvVejXfupdkv); vlzcDcTSrYXZiamsmNlx = 
(float*)malloc(eczxzisTMvVejXfupdkv); loadScale(LzeBIFvfQmAPApScLUyy); 
loadOffset(LvFwgsCgepJxzddrrZNH); 
loadTrainedMean(MXVlAOBGDMdzlrCMvAgl); 
loadTrainedVariance(MmeSJXZBVyUbgMuuFbuc); for (int i=0; 
i<ipTensor->getChannels(); i++) { qLXeoFROCbISdsnwpYgl[i] = 
pxmnUEWGnfCxJNuDkXAo[i]/sqrt(vlzcDcTSrYXZiamsmNlx[i] + RgALmBtPIZWDevjZBUHy); 
sXWXkiDEKpurgeCqZLDL[i] = hYTzvgWajqchLzrmxjqn[i] - 
(vcOGADqMrTrPPcuYvrHS[i]*qLXeoFROCbISdsnwpYgl[i]); 
niGnnRufksTFnsUUxnCj[i] = 1.f; } } qeQuIDaHqnxGPDbPoQJF.values = 
qLXeoFROCbISdsnwpYgl; qeQuIDaHqnxGPDbPoQJF.count = 
ipTensor->getChannels(); qeQuIDaHqnxGPDbPoQJF.type = DataType::kFLOAT; 
suFVgcuEVpCOrewbJfkB.values = sXWXkiDEKpurgeCqZLDL; 
suFVgcuEVpCOrewbJfkB.count = ipTensor->getChannels(); 
suFVgcuEVpCOrewbJfkB.type = DataType::kFLOAT; 
pKmXpiCPxZwpmXlulovZ.values = niGnnRufksTFnsUUxnCj; 
pKmXpiCPxZwpmXlulovZ.count = ipTensor->getChannels(); 
pKmXpiCPxZwpmXlulovZ.type = DataType::kFLOAT; ITensor* prevLayerTensor = 
getInputITensor(0); ATYqlAsSnRELrakAbCoK = 
eWYFXrUazhqiEIscccda->network->addScale(*prevLayerTensor, ScaleMode::kCHANNEL, 
suFVgcuEVpCOrewbJfkB, qeQuIDaHqnxGPDbPoQJF, 
pKmXpiCPxZwpmXlulovZ); 
ATYqlAsSnRELrakAbCoK->setName(getLayer()->getName().c_str());  
setOpTensorPtr(ATYqlAsSnRELrakAbCoK->getOutput(0)); } 
MWBatchNormalizationLayerImpl::~MWBatchNormalizationLayerImpl() { } void 
MWBatchNormalizationLayerImpl::iLoadParamOntoGPU(char const * const 
RuGYRQXjIMQJrbgoRUxZ, int const fdiBdaeFcIDdmsgMxaJT, float* 
TYgANfbwgYWWZKKtdxCC) { FILE* SZPsAnAecHGeFCSHofdG = 
MWCNNLayer::openBinaryFile(RuGYRQXjIMQJrbgoRUxZ); assert(SZPsAnAecHGeFCSHofdG); int 
const OwscQfaoXJuSJFwXQahz = sizeof(float)*fdiBdaeFcIDdmsgMxaJT; float* 
OWgntZrUmlZXHAsNObcq = (float*)malloc(OwscQfaoXJuSJFwXQahz); 
call_fread(OWgntZrUmlZXHAsNObcq, sizeof(float), fdiBdaeFcIDdmsgMxaJT, 
SZPsAnAecHGeFCSHofdG, RuGYRQXjIMQJrbgoRUxZ); fclose(SZPsAnAecHGeFCSHofdG); 
CUDA_CALL(hipMemcpy(TYgANfbwgYWWZKKtdxCC, OWgntZrUmlZXHAsNObcq, 
OwscQfaoXJuSJFwXQahz, hipMemcpyHostToDevice)); free(OWgntZrUmlZXHAsNObcq); } 
void MWBatchNormalizationLayerImpl::iLoadParam(char const * const 
RuGYRQXjIMQJrbgoRUxZ, int const fdiBdaeFcIDdmsgMxaJT, float* 
EMtxAWxHxCcPIkaNDIHM) { FILE* SZPsAnAecHGeFCSHofdG = 
MWCNNLayer::openBinaryFile(RuGYRQXjIMQJrbgoRUxZ); assert(SZPsAnAecHGeFCSHofdG); 
call_fread(EMtxAWxHxCcPIkaNDIHM, sizeof(float), fdiBdaeFcIDdmsgMxaJT, 
SZPsAnAecHGeFCSHofdG, RuGYRQXjIMQJrbgoRUxZ); fclose(SZPsAnAecHGeFCSHofdG); } void 
MWBatchNormalizationLayerImpl::loadScale(const char* RuGYRQXjIMQJrbgoRUxZ) { 
MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); if (QJJBjzDRkBQsCLkHaADa) 
iLoadParamOntoGPU(RuGYRQXjIMQJrbgoRUxZ, opTensor->getChannels(), 
pxmnUEWGnfCxJNuDkXAo); else iLoadParam(RuGYRQXjIMQJrbgoRUxZ, 
opTensor->getChannels(), pxmnUEWGnfCxJNuDkXAo); } void 
MWBatchNormalizationLayerImpl::loadOffset(const char* RuGYRQXjIMQJrbgoRUxZ) { 
MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); if (QJJBjzDRkBQsCLkHaADa) 
iLoadParamOntoGPU(RuGYRQXjIMQJrbgoRUxZ, opTensor->getChannels(), 
hYTzvgWajqchLzrmxjqn); else iLoadParam(RuGYRQXjIMQJrbgoRUxZ, 
opTensor->getChannels(), hYTzvgWajqchLzrmxjqn); } void 
MWBatchNormalizationLayerImpl::loadTrainedMean(const char* RuGYRQXjIMQJrbgoRUxZ) 
{ MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); if (QJJBjzDRkBQsCLkHaADa) 
iLoadParamOntoGPU(RuGYRQXjIMQJrbgoRUxZ, opTensor->getChannels(), 
vcOGADqMrTrPPcuYvrHS); else iLoadParam(RuGYRQXjIMQJrbgoRUxZ, 
opTensor->getChannels(), vcOGADqMrTrPPcuYvrHS); } void 
MWBatchNormalizationLayerImpl::loadTrainedVariance(const char* 
RuGYRQXjIMQJrbgoRUxZ) { MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); if (QJJBjzDRkBQsCLkHaADa) 
iLoadParamOntoGPU(RuGYRQXjIMQJrbgoRUxZ, opTensor->getChannels(), 
vlzcDcTSrYXZiamsmNlx); else iLoadParam(RuGYRQXjIMQJrbgoRUxZ, 
opTensor->getChannels(), vlzcDcTSrYXZiamsmNlx); } void 
MWBatchNormalizationLayerImpl::cleanup() { if 
(QJJBjzDRkBQsCLkHaADa) { hipFree(pxmnUEWGnfCxJNuDkXAo); 
hipFree(hYTzvgWajqchLzrmxjqn); hipFree(vcOGADqMrTrPPcuYvrHS); 
hipFree(vlzcDcTSrYXZiamsmNlx); } else { free(pxmnUEWGnfCxJNuDkXAo); 
free(hYTzvgWajqchLzrmxjqn); free(vcOGADqMrTrPPcuYvrHS); 
free(vlzcDcTSrYXZiamsmNlx); } free(qLXeoFROCbISdsnwpYgl); 
free(sXWXkiDEKpurgeCqZLDL); free(niGnnRufksTFnsUUxnCj); }