#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include "MWCNNLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t TAtTmiIniopalIZnGYzp; void 
curand_call_line_file(hiprandStatus_t tYWUxNVtgBrSjkBemGfF, const int 
aLQgzsOoQiSWjcCaRVKm, const char* RlwtKXlNBNTmWbDUiwqD) { if (tYWUxNVtgBrSjkBemGfF != 
HIPRAND_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"%d at line: %d, file: %s\n", tYWUxNVtgBrSjkBemGfF, aLQgzsOoQiSWjcCaRVKm, 
RlwtKXlNBNTmWbDUiwqD); throw std::runtime_error(buffer); } }
#endif
 void call_cuda_free(float* mem, const int aLQgzsOoQiSWjcCaRVKm, const char* 
RlwtKXlNBNTmWbDUiwqD) { if (!mem) { return; } hipError_t tYWUxNVtgBrSjkBemGfF = 
hipFree(mem); } float* malloc_call_line_file(size_t msize, const int 
aLQgzsOoQiSWjcCaRVKm, const char *RlwtKXlNBNTmWbDUiwqD) { float * mem = 
(float*)malloc(msize); if (!mem) { char buffer[100]; int numElem = 
sprintf(buffer, "%s at line: %d, file: %s\n", "Memory allocation failed. ", 
aLQgzsOoQiSWjcCaRVKm, RlwtKXlNBNTmWbDUiwqD); throw std::runtime_error(buffer); } return 
mem; } void cuda_call_line_file(hipError_t tYWUxNVtgBrSjkBemGfF, const int 
aLQgzsOoQiSWjcCaRVKm, const char* RlwtKXlNBNTmWbDUiwqD) { if (tYWUxNVtgBrSjkBemGfF != 
hipSuccess) { char buffer[100]; int numElem = sprintf(buffer, 
"Cuda Error %d(%s) at line: %d, file: %s\n", tYWUxNVtgBrSjkBemGfF, 
hipGetErrorString(tYWUxNVtgBrSjkBemGfF), aLQgzsOoQiSWjcCaRVKm, RlwtKXlNBNTmWbDUiwqD); 
tYWUxNVtgBrSjkBemGfF = hipGetLastError();  throw std::runtime_error(buffer); } } 
void cudnn_call_line_file(hipdnnStatus_t tYWUxNVtgBrSjkBemGfF, const int 
aLQgzsOoQiSWjcCaRVKm, const char* RlwtKXlNBNTmWbDUiwqD) { if (tYWUxNVtgBrSjkBemGfF != 
HIPDNN_STATUS_SUCCESS) {  char buffer[100]; int numElem = sprintf(buffer, 
"CuDNN Error %d(%s) at line: %d, file: %s\n", tYWUxNVtgBrSjkBemGfF, 
hipdnnGetErrorString(tYWUxNVtgBrSjkBemGfF), aLQgzsOoQiSWjcCaRVKm, RlwtKXlNBNTmWbDUiwqD); 
throw std::runtime_error(buffer); } } 
MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : aJTwGElOoWpBrmCfheqQ(layer) , eWYFXrUazhqiEIscccda(ntwk_impl) , 
RKrEonnJBdcnwoJXOHNM(0.0) , QlfGfPUqoazZMqFOfETJ(1.0) , QHUGvHzeHXyFElIiOliL(-1.0) , 
EiBytenrthqoQrTnOFaK(0) { } MWCNNLayerImpl::~MWCNNLayerImpl() { 
for(std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lxBZLYcHXoXUkMjfqsuo.begin(); it != lxBZLYcHXoXUkMjfqsuo.end(); ++it) { 
delete it->second; it->second = 0; } } ITensor* 
MWCNNLayerImpl::getInputITensor(int inputIdx) { MWTensor* ipTensor = 
getLayer()->getInputTensor(inputIdx); assert(ipTensor); return 
getITensor(ipTensor); } ITensor* MWCNNLayerImpl::getITensor(MWTensor* tensor) { 
if (tensor->getOwner()->getImpl() == NULL) { return 
getITensor(tensor->getOwner()->getInputTensor(0)); } else { return 
tensor->getOwner()->getImpl()->getOpTensorPtr(tensor->getSourcePortIndex()); } 
} hipdnnTensorDescriptor_t* MWCNNLayerImpl::getOutputDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lxBZLYcHXoXUkMjfqsuo.find(index); if (it == lxBZLYcHXoXUkMjfqsuo.end()) { 
hipdnnTensorDescriptor_t* tmp = new hipdnnTensorDescriptor_t; 
lxBZLYcHXoXUkMjfqsuo[index] = tmp; assert(tmp != 0); return tmp; } else { 
assert(it->second != 0); return it->second; } } void 
MWCNNLayerImpl::deallocateOutputData(){ for (int i = 0; i < 
getLayer()->getNumOutputs(); ++i){ MWTensor* opTensor = 
getLayer()->getOutputTensor(i); float* data = opTensor->getData<float>(); if 
(data) { CUDA_FREE_CALL(data); opTensor->setData((float*)NULL); } } } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getCuDNNDescriptor(MWTensor* tensor) { 
return tensor->getOwner()->getImpl()->getOutputDescriptor( 
tensor->getSourcePortIndex()); } int MWCNNLayerImpl::pluginEnqueueImpl(const 
void* const * , void** ){ assert(false); return 0; } 
MWPluginInterfaceImpl::MWPluginInterfaceImpl(MWCNNLayerImpl* 
PfNIOWjbRyfefiYoFSmL) : m_cnnLayerImpl(PfNIOWjbRyfefiYoFSmL){} Dims 
MWPluginInterfaceImpl::getOutputDimensions(int index, const Dims* , int ) { if 
(!m_cnnLayerImpl->eWYFXrUazhqiEIscccda->isSequenceNetwork){ int 
PIyXElJqMZoWKemWyTOa = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getChannels(); int 
TwiaHttwApyaipMEKPSg = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getHeight(); int 
znJVDnWdGXAXoBVlQhwT = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getWidth(); return 
DimsCHW(PIyXElJqMZoWKemWyTOa, TwiaHttwApyaipMEKPSg, znJVDnWdGXAXoBVlQhwT); }
#if (NV_TENSORRT_MAJOR >= 5)
 else{ int sPCEmfHYfjaRzyVvCKeA = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getSequenceLength(); int 
PIyXElJqMZoWKemWyTOa = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getChannels(); int 
NSzdekOvRhMhRCXdWsdY = 
m_cnnLayerImpl->getLayer()->getOutputTensor(index)->getBatchSize(); return 
Dims3(sPCEmfHYfjaRzyVvCKeA, NSzdekOvRhMhRCXdWsdY, PIyXElJqMZoWKemWyTOa); }
#endif 
 } void MWPluginInterfaceImpl::configure(const Dims* inputDims, int nbInputs, 
const Dims* outputDims, int nbOutputs, int ) { assert(inputDims->nbDims == 3);  
assert(outputDims->nbDims == 3);  assert(nbInputs == 
m_cnnLayerImpl->getLayer()->getNumInputs()); assert(nbOutputs == 
m_cnnLayerImpl->getLayer()->getNumOutputs()); } int 
MWPluginInterfaceImpl::getNbOutputs() const{ return 
m_cnnLayerImpl->getLayer()->getNumOutputs(); } int 
MWPluginInterfaceImpl::enqueue(int , const void* const* inputs, void** outputs, 
void* , hipStream_t ) { m_cnnLayerImpl->pluginEnqueueImpl(inputs,outputs); 
return 0; } MWInputLayerImpl::MWInputLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int dwFpvfypaTkJiYAULzFs, int TbrveedUYuqCPPSPaVab, int 
voqEJSkAwmNPuqzoiuom, int PAwKCndEJEByqwNZnPgb, int , const char* , int ) : 
MWCNNLayerImpl(layer, ntwk_impl) { MWTensor* opTensor = 
getLayer()->getOutputTensor(0); float * PsSZzscVKwYLIATdyqkh; 
CUDA_CALL(hipMalloc((void**)&PsSZzscVKwYLIATdyqkh, sizeof(float) * TbrveedUYuqCPPSPaVab * 
voqEJSkAwmNPuqzoiuom * PAwKCndEJEByqwNZnPgb * dwFpvfypaTkJiYAULzFs)); InputLayerITensor = 
eWYFXrUazhqiEIscccda->network->addInput( "data", DataType::kFLOAT, 
DimsCHW{PAwKCndEJEByqwNZnPgb, TbrveedUYuqCPPSPaVab, voqEJSkAwmNPuqzoiuom}); 
setOpTensorPtr(InputLayerITensor); opTensor->setData(PsSZzscVKwYLIATdyqkh); } void 
MWInputLayerImpl::cleanup() { for (int idx = 0; idx < 
aJTwGElOoWpBrmCfheqQ->getNumOutputs(); idx++) { float* data = 
aJTwGElOoWpBrmCfheqQ->getOutputTensor(idx)->getData<float>(); if (data) { 
CUDA_FREE_CALL(data); } } } MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int , int ) : MWCNNLayerImpl(layer, ntwk_impl) 
, iReLULayer(0) { ITensor* prevLayerTensor = getInputITensor(0); iReLULayer = 
eWYFXrUazhqiEIscccda->network->addActivation(*prevLayerTensor, 
ActivationType::kRELU); iReLULayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iReLULayer->getOutput(0)); } 
MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, unsigned IGBjAMvMJXqrubGDtvyq,  double AHTFZgpygljIqPClJcDZ,  
double AUjQjfbaYUcIYlesMFxV,  double EGziNPpAmkQdkYDEXfTU, int ) : MWCNNLayerImpl(layer, 
ntwk_impl) { ITensor* prevLayerTensor = getInputITensor(0); iNormLayer = 
eWYFXrUazhqiEIscccda->network->addLRN(*prevLayerTensor, 
IGBjAMvMJXqrubGDtvyq, AHTFZgpygljIqPClJcDZ, AUjQjfbaYUcIYlesMFxV, EGziNPpAmkQdkYDEXfTU); 
iNormLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iNormLayer->getOutput(0)); } void __global__ 
__launch_bounds__(1024) MWSetDyForBackPropImpl(float * RFQXHGHdWUKqrdBFLaiy, const int 
gxwFgFgfwoXAxqyOibKF) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
gxwFgFgfwoXAxqyOibKF; i+= blockDim.x*gridDim.x) { RFQXHGHdWUKqrdBFLaiy[i] = i+1; } } 
void __global__ __launch_bounds__(1024) doMWMaxPoolingLayerImpl(float * 
UROOthsHWeMcNycRifoq, float * UIgLxHHJdliWAJIeloVl, const int 
EOuFmpbshvhRMfQlfIXQ) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
EOuFmpbshvhRMfQlfIXQ; i+= blockDim.x*gridDim.x) { if 
(static_cast<int>(UROOthsHWeMcNycRifoq[i]) != 0){ 
UIgLxHHJdliWAJIeloVl[static_cast<int>(UROOthsHWeMcNycRifoq[i])-1] = 
i; } } } MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int GtFSKuDmLreppbjSISoU,  int HKStLBswJlYYprZPPGQx,  
int HhKGcPZwrclEFnIdWerH,  int HvFslbhtTwHuRNgeItfG, int GmRRxuYauzGdhIlgciAT, int 
FoMzPBFlspSYGUZHPvzd,  int GVFzDZAsZZUMIMwulTWX, int GbHRuweETkejIMGyqHDI, bool 
INKFbkrHldYkZFmALnfC, int MW_mangled_, const std::vector<int>& ) : 
MWCNNLayerImpl(layer, ntwk_impl) , iMaxPoolingLayer(0) , 
GrowsTaKrpHVUZdgZeJW(0) , mJnXzwDFPTieqFtWcZIG(0) , 
ThkGOmtrxiMfUeOSxFsN(INKFbkrHldYkZFmALnfC) { ITensor* prevLayerTensor = 
getInputITensor(0); if (!ThkGOmtrxiMfUeOSxFsN && (GmRRxuYauzGdhIlgciAT == 
FoMzPBFlspSYGUZHPvzd) && (GVFzDZAsZZUMIMwulTWX == GbHRuweETkejIMGyqHDI)){ 
iMaxPoolingLayer = eWYFXrUazhqiEIscccda->network->addPooling( *prevLayerTensor, 
PoolingType::kMAX, DimsHW{GtFSKuDmLreppbjSISoU, HKStLBswJlYYprZPPGQx}); 
iMaxPoolingLayer->setStride(DimsHW{HhKGcPZwrclEFnIdWerH, HvFslbhtTwHuRNgeItfG}); 
iMaxPoolingLayer->setPadding(DimsHW{GmRRxuYauzGdhIlgciAT, 
GVFzDZAsZZUMIMwulTWX}); 
iMaxPoolingLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iMaxPoolingLayer->getOutput(0)); } else{ 
pluginSetup(GtFSKuDmLreppbjSISoU, HKStLBswJlYYprZPPGQx, HhKGcPZwrclEFnIdWerH, 
HvFslbhtTwHuRNgeItfG, GmRRxuYauzGdhIlgciAT, GVFzDZAsZZUMIMwulTWX); 
mJnXzwDFPTieqFtWcZIG = new MWPluginInterfaceImpl(this); GrowsTaKrpHVUZdgZeJW = 
eWYFXrUazhqiEIscccda->network->addPlugin(&prevLayerTensor, 1, 
*mJnXzwDFPTieqFtWcZIG); setOpTensorPtr(GrowsTaKrpHVUZdgZeJW->getOutput(0),0); 
GrowsTaKrpHVUZdgZeJW->setName(getLayer()->getName().c_str()); if 
(ThkGOmtrxiMfUeOSxFsN) setOpTensorPtr(GrowsTaKrpHVUZdgZeJW->getOutput(1),1); 
} } float* MWMaxPoolingLayerImpl::getIndexData() { return NULL; } void 
MWMaxPoolingLayerImpl::cleanup() { if (mJnXzwDFPTieqFtWcZIG){ 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mrCdvmzPtAeVktINiAZK)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*XOJRvKzQwSaZobhyUoOi)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*jXNXIjpdcoiJUsfPyJJv)); } if 
(ThkGOmtrxiMfUeOSxFsN) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*kDIJsXmMuRtKrTNwutxt)); 
CUDA_FREE_CALL(UROOthsHWeMcNycRifoq); CUDA_FREE_CALL(RFQXHGHdWUKqrdBFLaiy); } 
} void MWMaxPoolingLayerImpl::pluginSetup(int GtFSKuDmLreppbjSISoU, int 
HKStLBswJlYYprZPPGQx, int HhKGcPZwrclEFnIdWerH, int HvFslbhtTwHuRNgeItfG, int 
GmRRxuYauzGdhIlgciAT, int GVFzDZAsZZUMIMwulTWX){ MWTensor* ipTensor = 
getLayer()->getInputTensor();  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mrCdvmzPtAeVktINiAZK)); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mrCdvmzPtAeVktINiAZK, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, GtFSKuDmLreppbjSISoU, HKStLBswJlYYprZPPGQx, 
GmRRxuYauzGdhIlgciAT, GVFzDZAsZZUMIMwulTWX, HhKGcPZwrclEFnIdWerH, 
HvFslbhtTwHuRNgeItfG)); XOJRvKzQwSaZobhyUoOi = new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(XOJRvKzQwSaZobhyUoOi)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*XOJRvKzQwSaZobhyUoOi, HIPDNN_TENSOR_NCHW,  
HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(),  ipTensor->getChannels(),  
ipTensor->getHeight(),  ipTensor->getWidth()));  int dwFpvfypaTkJiYAULzFs, 
PAwKCndEJEByqwNZnPgb, TbrveedUYuqCPPSPaVab, voqEJSkAwmNPuqzoiuom; 
CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(mrCdvmzPtAeVktINiAZK, 
*XOJRvKzQwSaZobhyUoOi, &dwFpvfypaTkJiYAULzFs ,&PAwKCndEJEByqwNZnPgb, &TbrveedUYuqCPPSPaVab, 
&voqEJSkAwmNPuqzoiuom)); TbrveedUYuqCPPSPaVab = getLayer()->getOutputTensor(0)->getHeight(); 
voqEJSkAwmNPuqzoiuom = getLayer()->getOutputTensor(0)->getWidth(); jXNXIjpdcoiJUsfPyJJv = 
new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(jXNXIjpdcoiJUsfPyJJv)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*jXNXIjpdcoiJUsfPyJJv, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, dwFpvfypaTkJiYAULzFs, PAwKCndEJEByqwNZnPgb, TbrveedUYuqCPPSPaVab, voqEJSkAwmNPuqzoiuom)); if 
(ThkGOmtrxiMfUeOSxFsN){ kDIJsXmMuRtKrTNwutxt = new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(kDIJsXmMuRtKrTNwutxt)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*kDIJsXmMuRtKrTNwutxt, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, dwFpvfypaTkJiYAULzFs, PAwKCndEJEByqwNZnPgb, TbrveedUYuqCPPSPaVab, voqEJSkAwmNPuqzoiuom)); 
assert((PAwKCndEJEByqwNZnPgb == ipTensor->getChannels()) && (dwFpvfypaTkJiYAULzFs == 
ipTensor->getBatchSize()));  fPIxBBGHjPkvmoaWByBr = 
(ipTensor->getHeight())*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
CUDA_CALL(hipMalloc((void**)&UROOthsHWeMcNycRifoq, 
sizeof(float)*fPIxBBGHjPkvmoaWByBr)); gxwFgFgfwoXAxqyOibKF = 
voqEJSkAwmNPuqzoiuom*TbrveedUYuqCPPSPaVab*PAwKCndEJEByqwNZnPgb*dwFpvfypaTkJiYAULzFs; 
CUDA_CALL(hipMalloc((void**)&RFQXHGHdWUKqrdBFLaiy, 
sizeof(float)*gxwFgFgfwoXAxqyOibKF)); int uqHugYAAqkSnCCYonqCt = 
(gxwFgFgfwoXAxqyOibKF < 1024) ? gxwFgFgfwoXAxqyOibKF : 1024; int 
OJTEGflbxqozjWWEaUJd = (gxwFgFgfwoXAxqyOibKF + uqHugYAAqkSnCCYonqCt - 
1)/uqHugYAAqkSnCCYonqCt; 
MWSetDyForBackPropImpl<<<OJTEGflbxqozjWWEaUJd, 
uqHugYAAqkSnCCYonqCt>>>( RFQXHGHdWUKqrdBFLaiy, gxwFgFgfwoXAxqyOibKF); } } int 
MWMaxPoolingLayerImpl::pluginEnqueueImpl(const void* const * inputs, void** 
outputs){ 
CUDNN_CALL(hipdnnPoolingForward(*eWYFXrUazhqiEIscccda->getCudnnHandle(), 
mrCdvmzPtAeVktINiAZK, getOnePtr(), *XOJRvKzQwSaZobhyUoOi, (float*)inputs[0], 
getZeroPtr(), *jXNXIjpdcoiJUsfPyJJv, (float*)outputs[0])); if 
(ThkGOmtrxiMfUeOSxFsN) { MWTensor* ipTensor = getLayer()->getInputTensor(); 
CUDNN_CALL(hipdnnPoolingBackward(*eWYFXrUazhqiEIscccda->getCudnnHandle(), 
mrCdvmzPtAeVktINiAZK, getOnePtr(), *jXNXIjpdcoiJUsfPyJJv, (float*)outputs[0], 
*jXNXIjpdcoiJUsfPyJJv, RFQXHGHdWUKqrdBFLaiy, *XOJRvKzQwSaZobhyUoOi, (float*)inputs[0], 
getZeroPtr(), *XOJRvKzQwSaZobhyUoOi, UROOthsHWeMcNycRifoq)); int 
uqHugYAAqkSnCCYonqCt = (fPIxBBGHjPkvmoaWByBr < 1024) ? fPIxBBGHjPkvmoaWByBr : 
1024; int OJTEGflbxqozjWWEaUJd = (fPIxBBGHjPkvmoaWByBr + 
uqHugYAAqkSnCCYonqCt - 1)/uqHugYAAqkSnCCYonqCt; 
doMWMaxPoolingLayerImpl<<<OJTEGflbxqozjWWEaUJd, 
uqHugYAAqkSnCCYonqCt>>>( UROOthsHWeMcNycRifoq, 
(float*)outputs[1], fPIxBBGHjPkvmoaWByBr); } return 0; } 
MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* ntwk_impl, 
int EfvWctmlsWAPsxXgdKWf, const char* zdBqSakTlqrPDQejGdMF,  const 
char* OFXGTgQYmVJLJPBNAVgS, int ) : MWCNNLayerImpl(layer, ntwk_impl) , 
iFCLayer(0) { MWTensor* opTensor = getLayer()->getOutputTensor(0); MWTensor* 
ipTensor = getLayer()->getInputTensor(0); voqEJSkAwmNPuqzoiuom = 
(float*)calloc(EfvWctmlsWAPsxXgdKWf * opTensor->getChannels(), 
sizeof(float)); NGqpeiLeVweDRsOKEtuw = (float*)calloc(opTensor->getChannels(), 
sizeof(float)); int eYGiuTCCxjmoBDvVpHpn = EfvWctmlsWAPsxXgdKWf * 
opTensor->getChannels();  loadWeights(eYGiuTCCxjmoBDvVpHpn, zdBqSakTlqrPDQejGdMF); 
loadBias(OFXGTgQYmVJLJPBNAVgS); ITensor* prevLayerITensor = getInputITensor(0); 
filt_weights.values = voqEJSkAwmNPuqzoiuom; filt_weights.count = 
EfvWctmlsWAPsxXgdKWf * opTensor->getChannels(); filt_weights.type = 
DataType::kFLOAT; filt_bias.values = NGqpeiLeVweDRsOKEtuw; filt_bias.count = 
opTensor->getChannels(); filt_bias.type = DataType::kFLOAT; if 
(!eWYFXrUazhqiEIscccda->isSequenceNetwork){ iFCLayer = 
eWYFXrUazhqiEIscccda->network->addFullyConnected( *prevLayerITensor, 
opTensor->getChannels(), filt_weights, filt_bias); 
iFCLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iFCLayer->getOutput(0)); }
#if (NV_TENSORRT_MAJOR >= 5)
 else{ auto shuffleLayer = 
eWYFXrUazhqiEIscccda->network->addShuffle(*prevLayerITensor); 
assert(shuffleLayer); shuffleLayer->setFirstTranspose(Permutation{1, 0, 2}); 
auto fcwts = eWYFXrUazhqiEIscccda->network->addConstant(Dims3(1, 
opTensor->getChannels(), EfvWctmlsWAPsxXgdKWf), filt_weights);
#if (NV_TENSORRT_MAJOR >= 5 && NV_TENSORRT_MINOR >= 1)
 auto matrixMultLayer = eWYFXrUazhqiEIscccda->network->addMatrixMultiply( 
*fcwts->getOutput(0), MatrixOperation::kNONE, *shuffleLayer->getOutput(0), MatrixOperation::kTRANSPOSE);
#else
 auto matrixMultLayer = eWYFXrUazhqiEIscccda->network->addMatrixMultiply( 
*fcwts->getOutput(0), false, *shuffleLayer->getOutput(0), true);
#endif
 assert(matrixMultLayer != nullptr); auto fcbias = 
eWYFXrUazhqiEIscccda->network->addConstant(Dims3(1, opTensor->getChannels(), 1), 
filt_bias); auto elementWiseLayer = 
eWYFXrUazhqiEIscccda->network->addElementWise(*matrixMultLayer->getOutput(0), 
*fcbias->getOutput(0), ElementWiseOperation::kSUM); assert(elementWiseLayer != 
nullptr); shuffleLayer = 
eWYFXrUazhqiEIscccda->network->addShuffle(*elementWiseLayer->getOutput(0)); 
assert(shuffleLayer); shuffleLayer->setFirstTranspose(Permutation{2, 0, 1}); 
setOpTensorPtr(shuffleLayer->getOutput(0)); }
#endif
 } void MWFCLayerImpl::loadWeights(int eYGiuTCCxjmoBDvVpHpn, const char* 
RtogJCavwOREhELwknZy) { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); FILE* SZPsAnAecHGeFCSHofdG = 
MWCNNLayer::openBinaryFile(RtogJCavwOREhELwknZy); assert(SZPsAnAecHGeFCSHofdG); 
call_fread(voqEJSkAwmNPuqzoiuom, sizeof(float), eYGiuTCCxjmoBDvVpHpn, SZPsAnAecHGeFCSHofdG, 
RtogJCavwOREhELwknZy); if (ipTensor->getHeight() != 1 && ipTensor->getWidth() != 
1) { float* OuTwywxKeMgznElXdjGp = (float*)malloc(sizeof(float) * 
ipTensor->getHeight() * ipTensor->getWidth()); for (int k = 0; k < 
eYGiuTCCxjmoBDvVpHpn / ipTensor->getHeight() / ipTensor->getWidth(); k++) { for (int 
i = 0; i < ipTensor->getHeight() * ipTensor->getWidth(); i++) { 
OuTwywxKeMgznElXdjGp[i] = voqEJSkAwmNPuqzoiuom[k * ipTensor->getHeight() * 
ipTensor->getWidth() + i]; } for (int j = 0; j < ipTensor->getHeight(); j++) 
for (int i = 0; i < ipTensor->getWidth(); i++) { voqEJSkAwmNPuqzoiuom[k * 
ipTensor->getHeight() * ipTensor->getWidth() + j * ipTensor->getWidth() + i] = 
OuTwywxKeMgznElXdjGp[j + i * ipTensor->getHeight()]; } } 
free(OuTwywxKeMgznElXdjGp); } fclose(SZPsAnAecHGeFCSHofdG); } void 
MWFCLayerImpl::loadBias(const char* RtogJCavwOREhELwknZy) { MWTensor* opTensor = 
getLayer()->getOutputTensor(0); FILE* SZPsAnAecHGeFCSHofdG = 
MWCNNLayer::openBinaryFile(RtogJCavwOREhELwknZy); assert(SZPsAnAecHGeFCSHofdG); int 
eYGiuTCCxjmoBDvVpHpn = opTensor->getChannels();  call_fread(NGqpeiLeVweDRsOKEtuw, 
sizeof(float), eYGiuTCCxjmoBDvVpHpn, SZPsAnAecHGeFCSHofdG, RtogJCavwOREhELwknZy); 
fclose(SZPsAnAecHGeFCSHofdG); } void MWFCLayerImpl::cleanup() { free(voqEJSkAwmNPuqzoiuom); 
free(NGqpeiLeVweDRsOKEtuw); } MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int ) : MWCNNLayerImpl(layer, ntwk_impl) , 
iSoftmaxLayer(0) { MWTensor* opTensor = getLayer()->getOutputTensor(0); 
ITensor* prevLayerTensor = getInputITensor(0); if 
(!eWYFXrUazhqiEIscccda->isSequenceNetwork){ iSoftmaxLayer = 
eWYFXrUazhqiEIscccda->network->addSoftMax(*prevLayerTensor); }
#if (NV_TENSORRT_MAJOR >= 5) 
 else{ iSoftmaxLayer = 
eWYFXrUazhqiEIscccda->network->addSoftMax(*prevLayerTensor); 
iSoftmaxLayer->setAxes(1<<2); }
#endif
 iSoftmaxLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iSoftmaxLayer->getOutput(0)); } 
MWOutputLayerImpl::MWOutputLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int ) : MWCNNLayerImpl(layer, ntwk_impl) { MWTensor* opTensor = 
getLayer()->getOutputTensor(0); float * PsSZzscVKwYLIATdyqkh; 
CUDA_CALL(hipMalloc((void**)&PsSZzscVKwYLIATdyqkh, sizeof(float) * 
opTensor->getNumElements())); ITensor* prevLayerTensor = getInputITensor(0); 
setOpTensorPtr(prevLayerTensor); opTensor->setData(PsSZzscVKwYLIATdyqkh); } void 
MWOutputLayerImpl::cleanup() { for (int idx = 0; idx < 
aJTwGElOoWpBrmCfheqQ->getNumOutputs(); idx++) { float* data = 
aJTwGElOoWpBrmCfheqQ->getOutputTensor(idx)->getData<float>(); if (data) { 
CUDA_FREE_CALL(data); } } } 
MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int GtFSKuDmLreppbjSISoU,  int HKStLBswJlYYprZPPGQx,  
int HhKGcPZwrclEFnIdWerH,  int HvFslbhtTwHuRNgeItfG,  int GmRRxuYauzGdhIlgciAT,  
int FoMzPBFlspSYGUZHPvzd,  int GVFzDZAsZZUMIMwulTWX, int GbHRuweETkejIMGyqHDI, 
int ) : MWCNNLayerImpl(layer, ntwk_impl) , iAvgPoolingLayer(0) { ITensor* 
prevLayerTensor = getInputITensor(0); if((GmRRxuYauzGdhIlgciAT == 
FoMzPBFlspSYGUZHPvzd) && (GVFzDZAsZZUMIMwulTWX == GbHRuweETkejIMGyqHDI)){  
iAvgPoolingLayer = eWYFXrUazhqiEIscccda->network->addPooling( *prevLayerTensor, 
PoolingType::kAVERAGE, DimsHW{GtFSKuDmLreppbjSISoU, HKStLBswJlYYprZPPGQx}); 
iAvgPoolingLayer->setPadding(DimsHW{GmRRxuYauzGdhIlgciAT, 
GVFzDZAsZZUMIMwulTWX}); } else { IPaddingLayer* iPaddingLayer = 
eWYFXrUazhqiEIscccda->network->addPadding( *prevLayerTensor, 
DimsHW{GmRRxuYauzGdhIlgciAT,GVFzDZAsZZUMIMwulTWX}, 
DimsHW{FoMzPBFlspSYGUZHPvzd,GbHRuweETkejIMGyqHDI}); ITensor* 
EpwuhXsRcwdqXSjBpUeO = iPaddingLayer->getOutput(0); iAvgPoolingLayer = 
eWYFXrUazhqiEIscccda->network->addPooling( *EpwuhXsRcwdqXSjBpUeO, 
PoolingType::kAVERAGE, DimsHW{GtFSKuDmLreppbjSISoU, HKStLBswJlYYprZPPGQx});  } 
iAvgPoolingLayer->setStride(DimsHW{HhKGcPZwrclEFnIdWerH, HvFslbhtTwHuRNgeItfG}); 
iAvgPoolingLayer->setAverageCountExcludesPadding(false); 
iAvgPoolingLayer->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(iAvgPoolingLayer->getOutput(0)); }