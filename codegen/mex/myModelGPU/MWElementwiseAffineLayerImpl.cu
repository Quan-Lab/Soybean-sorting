#include "MWElementwiseAffineLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWKernelHeaders.hpp"
 MWElementwiseAffineLayerImpl::MWElementwiseAffineLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int scale_H,  int scale_W,  int scale_C, int 
offset_H,  int offset_W,  int offset_C, bool isClipped,  int lowerbound,  int 
upperbound, const char* scale_file,  const char* offset_file, int ) : 
MWCNNLayerImpl(layer, ntwk_impl), pxmnUEWGnfCxJNuDkXAo(NULL), 
hYTzvgWajqchLzrmxjqn(NULL), rZyMIPooLjRiXLgSWDuw(scale_H), 
rwPhFWHcKnJsClVtebGW(scale_W), qquNiJHQtfSLDMNCPIBJ(scale_C), 
hqVFaqkobRNLQNgtbaai(offset_H), ikTyjLTPRBkBRlLSyxXG(offset_W), 
hpOzCTZasBMYKoXVxMDZ(offset_C), ZqQxEyCjEixByRZYMkbj(isClipped), 
crKSAZwnyiinNFYODxoN(lowerbound), vmBqKEmdajzGggqevoGl(upperbound), 
sCDdEyIOjXBVHhcakBhd(nullptr), jLmklYtHcmTxayQTpmRw(nullptr), 
qJWXFXvcpbSwehmlTNru(0), GrowsTaKrpHVUZdgZeJW(0), mJnXzwDFPTieqFtWcZIG(0) { 
loadScaleAndOffset(scale_file, offset_file); setLayerProperties(); bool 
isMatrix2d = (rZyMIPooLjRiXLgSWDuw > 1) && (rwPhFWHcKnJsClVtebGW > 1) && 
(qquNiJHQtfSLDMNCPIBJ != WawamKKnqecNqBXIyHIl); if ((!ZqQxEyCjEixByRZYMkbj) && 
(reGtUwUlPSwEenEBVIzH == hqbKXLMjsDxRQqyJEgbg ) && !isMatrix2d && 
(!eWYFXrUazhqiEIscccda->isSequenceNetwork)) { qeQuIDaHqnxGPDbPoQJF.values 
= sCDdEyIOjXBVHhcakBhd; qeQuIDaHqnxGPDbPoQJF.count = reGtUwUlPSwEenEBVIzH; 
qeQuIDaHqnxGPDbPoQJF.type = DataType::kFLOAT; 
pKmXpiCPxZwpmXlulovZ.values = nullptr; pKmXpiCPxZwpmXlulovZ.count = 
0; pKmXpiCPxZwpmXlulovZ.type = DataType::kFLOAT; 
suFVgcuEVpCOrewbJfkB.values = jLmklYtHcmTxayQTpmRw; 
suFVgcuEVpCOrewbJfkB.count = hqbKXLMjsDxRQqyJEgbg; 
suFVgcuEVpCOrewbJfkB.type = DataType::kFLOAT; ITensor* prevLayerTensor = 
getInputITensor(0); ScaleMode mode; if (reGtUwUlPSwEenEBVIzH == 1) mode = 
ScaleMode::kUNIFORM; else if (YMNbgnUYZspjMLjwcIOS == 
reGtUwUlPSwEenEBVIzH) mode = ScaleMode::kELEMENTWISE; else if (rZyMIPooLjRiXLgSWDuw 
== 1 && rwPhFWHcKnJsClVtebGW == 1 && reGtUwUlPSwEenEBVIzH == qquNiJHQtfSLDMNCPIBJ) 
mode = ScaleMode::kCHANNEL; qJWXFXvcpbSwehmlTNru = 
eWYFXrUazhqiEIscccda->network->addScale(*prevLayerTensor,  mode,  
suFVgcuEVpCOrewbJfkB, qeQuIDaHqnxGPDbPoQJF,  
pKmXpiCPxZwpmXlulovZ); assert(qJWXFXvcpbSwehmlTNru); 
qJWXFXvcpbSwehmlTNru->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(qJWXFXvcpbSwehmlTNru->getOutput(0)); } else { ITensor* 
prevLayerTensor = getInputITensor(0); mJnXzwDFPTieqFtWcZIG = new 
MWPluginInterfaceImpl(this); GrowsTaKrpHVUZdgZeJW = 
eWYFXrUazhqiEIscccda->network->addPlugin(&prevLayerTensor, 1, 
*mJnXzwDFPTieqFtWcZIG); 
GrowsTaKrpHVUZdgZeJW->setName(getLayer()->getName().c_str()); 
setOpTensorPtr(GrowsTaKrpHVUZdgZeJW->getOutput(0)); } } void 
MWElementwiseAffineLayerImpl::loadScaleAndOffset(const char* 
sDWnRjToSPjYnOQzVfhS, const char* jNxFsuLXTFYGOUlfRwLW){ 
CUDA_CALL(hipMalloc((void**)&pxmnUEWGnfCxJNuDkXAo, 
sizeof(float)*rZyMIPooLjRiXLgSWDuw*rwPhFWHcKnJsClVtebGW*qquNiJHQtfSLDMNCPIBJ)); 
CUDA_CALL(hipMalloc((void**)&hYTzvgWajqchLzrmxjqn, 
sizeof(float)*hqVFaqkobRNLQNgtbaai*ikTyjLTPRBkBRlLSyxXG*hpOzCTZasBMYKoXVxMDZ));  
loadScale(sDWnRjToSPjYnOQzVfhS); loadOffset(jNxFsuLXTFYGOUlfRwLW); } void 
MWElementwiseAffineLayerImpl::setLayerProperties(){ WbTBQxsNsCURmwRhNTAD = 
getLayer()->getInputTensor(0)->getHeight(); XGQjNlvPuckcHnviTrkP = 
getLayer()->getInputTensor(0)->getWidth(); WawamKKnqecNqBXIyHIl = 
getLayer()->getInputTensor(0)->getChannels(); YmfPcXPXNFZDznkzKZrl = 
WbTBQxsNsCURmwRhNTAD*XGQjNlvPuckcHnviTrkP; YMNbgnUYZspjMLjwcIOS = 
YmfPcXPXNFZDznkzKZrl*WawamKKnqecNqBXIyHIl; YDoginwuwFxabuYCVqpT = 
getLayer()->getInputTensor(0)->getNumElements(); reGtUwUlPSwEenEBVIzH = 
rZyMIPooLjRiXLgSWDuw * rwPhFWHcKnJsClVtebGW * qquNiJHQtfSLDMNCPIBJ; 
hqbKXLMjsDxRQqyJEgbg = hqVFaqkobRNLQNgtbaai * ikTyjLTPRBkBRlLSyxXG * 
hpOzCTZasBMYKoXVxMDZ; assert(reGtUwUlPSwEenEBVIzH <= YDoginwuwFxabuYCVqpT); 
assert(hqbKXLMjsDxRQqyJEgbg <= YDoginwuwFxabuYCVqpT); } int 
MWElementwiseAffineLayerImpl::pluginEnqueueImpl(const void* const* inputs, 
void** outputs) { long int uTUuLVVebDakbPjXOQwp = ((YDoginwuwFxabuYCVqpT + 31) / 32) 
* 32; long int uqHugYAAqkSnCCYonqCt = (uTUuLVVebDakbPjXOQwp < 1024) ? 
uTUuLVVebDakbPjXOQwp : 1024; long int OJTEGflbxqozjWWEaUJd = 
(YDoginwuwFxabuYCVqpT + uqHugYAAqkSnCCYonqCt - 1) / 
uqHugYAAqkSnCCYonqCt; if (reGtUwUlPSwEenEBVIzH == 1) { 
scale_scalar_kernel<<<OJTEGflbxqozjWWEaUJd, uqHugYAAqkSnCCYonqCt>>>( 
(float*)inputs[0],  (float*)outputs[0], pxmnUEWGnfCxJNuDkXAo, 
YDoginwuwFxabuYCVqpT); } else if (rZyMIPooLjRiXLgSWDuw == 1 && rwPhFWHcKnJsClVtebGW 
== 1 && reGtUwUlPSwEenEBVIzH > 1) { 
scale_vector_kernel<<<OJTEGflbxqozjWWEaUJd, uqHugYAAqkSnCCYonqCt>>>( 
(float*)inputs[0],  (float*)outputs[0], pxmnUEWGnfCxJNuDkXAo, 
YmfPcXPXNFZDznkzKZrl, YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT); } else if (YMNbgnUYZspjMLjwcIOS == 
reGtUwUlPSwEenEBVIzH) {  scale_tensor3d_kernel<<<OJTEGflbxqozjWWEaUJd, 
uqHugYAAqkSnCCYonqCt>>>( (float*)inputs[0],  (float*)outputs[0], 
pxmnUEWGnfCxJNuDkXAo, XGQjNlvPuckcHnviTrkP, WbTBQxsNsCURmwRhNTAD,  
YmfPcXPXNFZDznkzKZrl,  YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT); } else { 
scale_matrix2d_kernel<<<OJTEGflbxqozjWWEaUJd, 
uqHugYAAqkSnCCYonqCt>>>( (float*)inputs[0],  (float*)outputs[0], 
pxmnUEWGnfCxJNuDkXAo, XGQjNlvPuckcHnviTrkP,  YmfPcXPXNFZDznkzKZrl,  
YMNbgnUYZspjMLjwcIOS, YDoginwuwFxabuYCVqpT); } if (hqbKXLMjsDxRQqyJEgbg 
== 1) { offset_scalar_kernel<<<OJTEGflbxqozjWWEaUJd, 
uqHugYAAqkSnCCYonqCt>>>( (float*)outputs[0],  (float*)outputs[0], 
hYTzvgWajqchLzrmxjqn, YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, 
crKSAZwnyiinNFYODxoN, vmBqKEmdajzGggqevoGl); } else if (hqVFaqkobRNLQNgtbaai 
== 1 && ikTyjLTPRBkBRlLSyxXG == 1 && hqbKXLMjsDxRQqyJEgbg > 1) { 
offset_vector_kernel<<<OJTEGflbxqozjWWEaUJd, uqHugYAAqkSnCCYonqCt>>>( 
(float*)outputs[0],  (float*)outputs[0], hYTzvgWajqchLzrmxjqn, 
YmfPcXPXNFZDznkzKZrl, YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, crKSAZwnyiinNFYODxoN, 
vmBqKEmdajzGggqevoGl); } else if (YMNbgnUYZspjMLjwcIOS == 
hqbKXLMjsDxRQqyJEgbg) { offset_tensor3d_kernel<<<OJTEGflbxqozjWWEaUJd, 
uqHugYAAqkSnCCYonqCt>>>( (float*)outputs[0],  (float*)outputs[0], 
hYTzvgWajqchLzrmxjqn, XGQjNlvPuckcHnviTrkP, WbTBQxsNsCURmwRhNTAD, 
YmfPcXPXNFZDznkzKZrl, YMNbgnUYZspjMLjwcIOS, 
YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, crKSAZwnyiinNFYODxoN, 
vmBqKEmdajzGggqevoGl); } else { 
offset_matrix2d_kernel<<<OJTEGflbxqozjWWEaUJd, 
uqHugYAAqkSnCCYonqCt>>>( (float*)outputs[0],  (float*)outputs[0], 
hYTzvgWajqchLzrmxjqn, XGQjNlvPuckcHnviTrkP, YmfPcXPXNFZDznkzKZrl, 
YMNbgnUYZspjMLjwcIOS, YDoginwuwFxabuYCVqpT, ZqQxEyCjEixByRZYMkbj, 
crKSAZwnyiinNFYODxoN, vmBqKEmdajzGggqevoGl); } return 0; } void 
MWElementwiseAffineLayerImpl::loadScale(const char* sDWnRjToSPjYnOQzVfhS) { 
FILE* SZPsAnAecHGeFCSHofdG = MWCNNLayer::openBinaryFile(sDWnRjToSPjYnOQzVfhS); 
assert(SZPsAnAecHGeFCSHofdG); long int eYGiuTCCxjmoBDvVpHpn = 
rZyMIPooLjRiXLgSWDuw*rwPhFWHcKnJsClVtebGW*qquNiJHQtfSLDMNCPIBJ; sCDdEyIOjXBVHhcakBhd 
= MALLOC_CALL(sizeof(float)*eYGiuTCCxjmoBDvVpHpn); call_fread(sCDdEyIOjXBVHhcakBhd, 
sizeof(float), eYGiuTCCxjmoBDvVpHpn, SZPsAnAecHGeFCSHofdG, sDWnRjToSPjYnOQzVfhS); 
CUDA_CALL(hipMemcpy(pxmnUEWGnfCxJNuDkXAo, sCDdEyIOjXBVHhcakBhd, 
sizeof(float)*eYGiuTCCxjmoBDvVpHpn, hipMemcpyHostToDevice)); fclose(SZPsAnAecHGeFCSHofdG);  
} void MWElementwiseAffineLayerImpl::loadOffset(const char* 
jNxFsuLXTFYGOUlfRwLW) { FILE* SZPsAnAecHGeFCSHofdG = 
MWCNNLayer::openBinaryFile(jNxFsuLXTFYGOUlfRwLW); assert(SZPsAnAecHGeFCSHofdG); long 
int eYGiuTCCxjmoBDvVpHpn = 
hqVFaqkobRNLQNgtbaai*ikTyjLTPRBkBRlLSyxXG*hpOzCTZasBMYKoXVxMDZ; 
jLmklYtHcmTxayQTpmRw = MALLOC_CALL(sizeof(float)*eYGiuTCCxjmoBDvVpHpn); 
call_fread(jLmklYtHcmTxayQTpmRw, sizeof(float), eYGiuTCCxjmoBDvVpHpn, SZPsAnAecHGeFCSHofdG, 
jNxFsuLXTFYGOUlfRwLW); CUDA_CALL(hipMemcpy(hYTzvgWajqchLzrmxjqn, 
jLmklYtHcmTxayQTpmRw, sizeof(float)*eYGiuTCCxjmoBDvVpHpn, hipMemcpyHostToDevice)); 
fclose(SZPsAnAecHGeFCSHofdG);  } void MWElementwiseAffineLayerImpl::cleanup() { if 
(pxmnUEWGnfCxJNuDkXAo) { CUDA_FREE_CALL(pxmnUEWGnfCxJNuDkXAo); } if (hYTzvgWajqchLzrmxjqn) 
{ CUDA_FREE_CALL(hYTzvgWajqchLzrmxjqn); } if (sCDdEyIOjXBVHhcakBhd) 
free(sCDdEyIOjXBVHhcakBhd); if (jLmklYtHcmTxayQTpmRw) 
free(jLmklYtHcmTxayQTpmRw); }