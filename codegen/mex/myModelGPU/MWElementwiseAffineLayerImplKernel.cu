#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <stdio.h>
 void __global__ __launch_bounds__(1024) scale_scalar_kernel(float* 
inputBuffer, float* outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int 
YNmJhGSUszJKxsodxiuV) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) {  outputBuffer[idx] 
= omxlPZbBePZdWaJOBUUG[0]*inputBuffer[idx]; } } void __global__ 
__launch_bounds__(1024) scale_vector_kernel(float* inputBuffer, float* 
outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int YeIFysyIXePEVfpcANol, 
long int YOWMnLKOMqAODXiVNoGy, long int YNmJhGSUszJKxsodxiuV) {  for 
(int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; 
idx += blockDim.x * gridDim.x) { int dMxIKDGTITyhdLqIHBLA = 
idx/YOWMnLKOMqAODXiVNoGy; long int FLuSVNoPhAFKtLUchSvv = 
idx-(YOWMnLKOMqAODXiVNoGy*dMxIKDGTITyhdLqIHBLA); int LgxABSJPBXdCozJkFqTg = 
static_cast<int>(FLuSVNoPhAFKtLUchSvv / YeIFysyIXePEVfpcANol); 
outputBuffer[idx] = omxlPZbBePZdWaJOBUUG[LgxABSJPBXdCozJkFqTg]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_matrix2d_kernel(float* inputBuffer, 
float* outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int YgcpEBUCwCLaPhyntIio, long 
int YeIFysyIXePEVfpcANol, long int YOWMnLKOMqAODXiVNoGy, long 
int YNmJhGSUszJKxsodxiuV) {  for (int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { int 
dMxIKDGTITyhdLqIHBLA = idx/YOWMnLKOMqAODXiVNoGy; long int FLuSVNoPhAFKtLUchSvv 
= idx-(YOWMnLKOMqAODXiVNoGy*dMxIKDGTITyhdLqIHBLA); int LgxABSJPBXdCozJkFqTg = 
static_cast<int>(FLuSVNoPhAFKtLUchSvv / YeIFysyIXePEVfpcANol); long 
int FOcStuqCptsGIZXskVpC = FLuSVNoPhAFKtLUchSvv - 
(YeIFysyIXePEVfpcANol*LgxABSJPBXdCozJkFqTg); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(FOcStuqCptsGIZXskVpC % YgcpEBUCwCLaPhyntIio); int 
RVrPByQXdKmunRZHKWJD = static_cast<int>(FOcStuqCptsGIZXskVpC / YgcpEBUCwCLaPhyntIio); 
outputBuffer[idx] = 
omxlPZbBePZdWaJOBUUG[vIWQzNvYZSuxmOTVDFhU+YgcpEBUCwCLaPhyntIio*RVrPByQXdKmunRZHKWJD]*inputBuffer[idx]; 
} } void __global__ __launch_bounds__(1024) scale_tensor3d_kernel(float* 
inputBuffer, float* outputBuffer, float* omxlPZbBePZdWaJOBUUG, long int 
YgcpEBUCwCLaPhyntIio, long int YGiQICncmsGZkNUyiQyg, long int 
YeIFysyIXePEVfpcANol, long int YOWMnLKOMqAODXiVNoGy, long int 
YNmJhGSUszJKxsodxiuV) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { int dMxIKDGTITyhdLqIHBLA = 
idx/YOWMnLKOMqAODXiVNoGy; long int FLuSVNoPhAFKtLUchSvv = 
idx-(YOWMnLKOMqAODXiVNoGy*dMxIKDGTITyhdLqIHBLA); int LgxABSJPBXdCozJkFqTg = 
static_cast<int>(FLuSVNoPhAFKtLUchSvv / YeIFysyIXePEVfpcANol); long 
int FOcStuqCptsGIZXskVpC = FLuSVNoPhAFKtLUchSvv - 
(YeIFysyIXePEVfpcANol*LgxABSJPBXdCozJkFqTg); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(FOcStuqCptsGIZXskVpC % YgcpEBUCwCLaPhyntIio); int 
RVrPByQXdKmunRZHKWJD = static_cast<int>(FOcStuqCptsGIZXskVpC / YgcpEBUCwCLaPhyntIio); 
outputBuffer[idx] = 
omxlPZbBePZdWaJOBUUG[vIWQzNvYZSuxmOTVDFhU+YgcpEBUCwCLaPhyntIio*(RVrPByQXdKmunRZHKWJD+YGiQICncmsGZkNUyiQyg*LgxABSJPBXdCozJkFqTg)]*inputBuffer[idx]; 
} }  void __global__ __launch_bounds__(1024) offset_scalar_kernel(float* 
inputBuffer, float* outputBuffer, float* gTcJMwtYuwiqqUmqvKhT, long int 
YNmJhGSUszJKxsodxiuV, bool ZinudJuZuGitiNTsJpBR, int bUVPfnrJhLfHzOLUUrKk, int 
unSXtdjDjpysqxmbIiPv) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { outputBuffer[idx] 
= inputBuffer[idx] + gTcJMwtYuwiqqUmqvKhT[0]; if (ZinudJuZuGitiNTsJpBR){ 
outputBuffer[idx] = outputBuffer[idx] > unSXtdjDjpysqxmbIiPv ? 
unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; outputBuffer[idx] = 
outputBuffer[idx] < bUVPfnrJhLfHzOLUUrKk ? bUVPfnrJhLfHzOLUUrKk : 
outputBuffer[idx]; } } } void __global__ __launch_bounds__(1024) 
offset_vector_kernel(float* inputBuffer, float* outputBuffer, float* 
gTcJMwtYuwiqqUmqvKhT,  long int YeIFysyIXePEVfpcANol, long int 
YOWMnLKOMqAODXiVNoGy, long int YNmJhGSUszJKxsodxiuV, bool 
ZinudJuZuGitiNTsJpBR, int bUVPfnrJhLfHzOLUUrKk, int unSXtdjDjpysqxmbIiPv) {  
for (int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { int dMxIKDGTITyhdLqIHBLA = 
idx/YOWMnLKOMqAODXiVNoGy; long int FLuSVNoPhAFKtLUchSvv = 
idx-(YOWMnLKOMqAODXiVNoGy*dMxIKDGTITyhdLqIHBLA); int LgxABSJPBXdCozJkFqTg = 
static_cast<int>(FLuSVNoPhAFKtLUchSvv / YeIFysyIXePEVfpcANol); 
outputBuffer[idx] = inputBuffer[idx] + gTcJMwtYuwiqqUmqvKhT[LgxABSJPBXdCozJkFqTg]; if 
(ZinudJuZuGitiNTsJpBR){ outputBuffer[idx] = outputBuffer[idx] > 
unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; 
outputBuffer[idx] = outputBuffer[idx] < bUVPfnrJhLfHzOLUUrKk ? 
bUVPfnrJhLfHzOLUUrKk : outputBuffer[idx]; } } } void __global__ 
__launch_bounds__(1024) offset_matrix2d_kernel(float* inputBuffer, float* 
outputBuffer, float* gTcJMwtYuwiqqUmqvKhT, long int YgcpEBUCwCLaPhyntIio, long int 
YeIFysyIXePEVfpcANol, long int YOWMnLKOMqAODXiVNoGy, long int 
YNmJhGSUszJKxsodxiuV, bool ZinudJuZuGitiNTsJpBR, int bUVPfnrJhLfHzOLUUrKk, int 
unSXtdjDjpysqxmbIiPv) {  for (int idx = blockDim.x * blockIdx.x + threadIdx.x; 
idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { int dMxIKDGTITyhdLqIHBLA = 
idx/YOWMnLKOMqAODXiVNoGy; long int FLuSVNoPhAFKtLUchSvv = 
idx-(YOWMnLKOMqAODXiVNoGy*dMxIKDGTITyhdLqIHBLA); int LgxABSJPBXdCozJkFqTg = 
static_cast<int>(FLuSVNoPhAFKtLUchSvv / YeIFysyIXePEVfpcANol); long 
int FOcStuqCptsGIZXskVpC = FLuSVNoPhAFKtLUchSvv - 
(YeIFysyIXePEVfpcANol*LgxABSJPBXdCozJkFqTg); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(FOcStuqCptsGIZXskVpC % YgcpEBUCwCLaPhyntIio); int 
RVrPByQXdKmunRZHKWJD = static_cast<int>(FOcStuqCptsGIZXskVpC / YgcpEBUCwCLaPhyntIio); 
outputBuffer[idx] = inputBuffer[idx] + 
gTcJMwtYuwiqqUmqvKhT[vIWQzNvYZSuxmOTVDFhU+YgcpEBUCwCLaPhyntIio*RVrPByQXdKmunRZHKWJD]; if 
(ZinudJuZuGitiNTsJpBR){ outputBuffer[idx] = outputBuffer[idx] > 
unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; 
outputBuffer[idx] = outputBuffer[idx] < bUVPfnrJhLfHzOLUUrKk ? 
bUVPfnrJhLfHzOLUUrKk : outputBuffer[idx]; } } } void __global__ 
__launch_bounds__(1024) offset_tensor3d_kernel(float* inputBuffer, float* 
outputBuffer, float* gTcJMwtYuwiqqUmqvKhT,  long int YgcpEBUCwCLaPhyntIio, long int 
YGiQICncmsGZkNUyiQyg, long int YeIFysyIXePEVfpcANol, long int 
YOWMnLKOMqAODXiVNoGy, long int YNmJhGSUszJKxsodxiuV, bool 
ZinudJuZuGitiNTsJpBR, int bUVPfnrJhLfHzOLUUrKk, int unSXtdjDjpysqxmbIiPv) {  
for (int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { int dMxIKDGTITyhdLqIHBLA = 
idx/YOWMnLKOMqAODXiVNoGy; long int FLuSVNoPhAFKtLUchSvv = 
idx-(YOWMnLKOMqAODXiVNoGy*dMxIKDGTITyhdLqIHBLA); int LgxABSJPBXdCozJkFqTg = 
static_cast<int>(FLuSVNoPhAFKtLUchSvv / YeIFysyIXePEVfpcANol); long 
int FOcStuqCptsGIZXskVpC = FLuSVNoPhAFKtLUchSvv - 
(YeIFysyIXePEVfpcANol*LgxABSJPBXdCozJkFqTg); int vIWQzNvYZSuxmOTVDFhU = 
static_cast<int>(FOcStuqCptsGIZXskVpC % YgcpEBUCwCLaPhyntIio); int 
RVrPByQXdKmunRZHKWJD = static_cast<int>(FOcStuqCptsGIZXskVpC / YgcpEBUCwCLaPhyntIio); 
outputBuffer[idx] = inputBuffer[idx] + 
gTcJMwtYuwiqqUmqvKhT[vIWQzNvYZSuxmOTVDFhU+YgcpEBUCwCLaPhyntIio*(RVrPByQXdKmunRZHKWJD+YGiQICncmsGZkNUyiQyg*LgxABSJPBXdCozJkFqTg)]; 
if (ZinudJuZuGitiNTsJpBR){ outputBuffer[idx] = outputBuffer[idx] > 
unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : outputBuffer[idx]; 
outputBuffer[idx] = outputBuffer[idx] < bUVPfnrJhLfHzOLUUrKk ? 
bUVPfnrJhLfHzOLUUrKk : outputBuffer[idx]; } } } 