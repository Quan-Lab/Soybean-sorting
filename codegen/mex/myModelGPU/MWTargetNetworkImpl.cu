#include "hip/hip_runtime.h"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWCNNLayerImpl.hpp"
#include <math.h>
#include <iostream>
#include <cassert>
#include <fstream>
#if INT8_ENABLED
#include <fstream>
#include <iterator>
#include "MWBatchStream.hpp"
#define XSTR(x) #x
#define STR(x) XSTR(x)
#endif
 using namespace nvinfer1; using namespace nvcaffeparser1; void 
CHECK(hipError_t status) { if (status != 0) { std::cout << "Cuda failure: " << 
status; abort(); } } class Logger : public ILogger { void log(Severity 
severity, const char* msg) override { if (severity != Severity::kINFO){ 
std::cout << msg << std::endl; } if (MWCNNLayer::isDebuggingEnabled()){  if 
(severity == Severity::kINFO){ std::ofstream logfile; 
logfile.open("MW_TensorRT_log.txt" , std::ofstream::out | std::ofstream::app); 
logfile << msg <<"\n"; logfile.close(); } } } }; static Logger gLogger;
#if INT8_ENABLED
 std::string getFilePath(std:: string fileS, std::string &path) { char* 
usrDataPath = NULL; usrDataPath = getenv("USER_DL_DATA_PATH"); if(usrDataPath 
!= NULL) { path = usrDataPath; } else { path = STR(MW_DL_DATA_PATH); } path = 
path + "/tensorrt"; size_t fNamePos = fileS.find_last_of("/\\"); if(fNamePos != 
std::string::npos) { std::string fileN(fileS.substr(fNamePos)); fileS = path + 
fileN; } else { fileS = path + fileS; } return fileS; } std::string 
gvalidDatapath;  void getValidDataPath(const char* fileName, char 
*validDatapath) { FILE* fp = fopen(fileName, "rb"); std::string 
fileS(fileName); if (!fp) {
#ifdef MW_DL_DATA_PATH
 std::string path; fileS = getFilePath(fileS,path); fp = fopen(fileS.c_str(), 
"rb"); if(fp != NULL) { fclose(fp); gvalidDatapath = path; 
strcpy(validDatapath,fileS.c_str()); } else { strcpy(validDatapath,fileName); }
#else
 size_t pos = 0;
#if defined(_WIN32) || defined(_WIN64)
 char delim_unix[] = "/"; char delim_win[] = "\\"; while(((pos = 
fileS.find(delim_unix)) != std::string::npos) || ((pos = fileS.find(delim_win)) 
!= std::string::npos))
#else
 char delim_unix[] = "/"; while((pos = fileS.find(delim_unix)) != std::string::npos)
#endif
 { if (pos == (fileS.size() - 1)) { fileS = ""; break; } fileS = 
fileS.substr(pos+1); fp = fopen(fileS.c_str(), "rb"); if(fp != NULL) { 
fclose(fp); strcpy(validDatapath, fileS.c_str()); gvalidDatapath = 
fileS.substr(0,fileS.find_last_of("/\\")); break; } else{ strcpy(validDatapath, 
fileName); } }
#endif
 } else { fclose(fp); strcpy(validDatapath, fileName); gvalidDatapath 
=validDatapath; gvalidDatapath = 
gvalidDatapath.substr(0,gvalidDatapath.find_last_of("/\\")); } }
#endif
 void MWTargetNetworkImpl::setBatchSize(int aBatchSize){ batchSize = 
aBatchSize; } void MWTargetNetworkImpl::setIsSequenceNetwork(bool 
aIsSequenceNetwork){ isSequenceNetwork = aIsSequenceNetwork; } void 
MWTargetNetworkImpl::doInference(int batchSize) { const ICudaEngine& engine = 
context->getEngine(); hipStream_t stream; CHECK(hipStreamCreate(&stream)); if 
(this->isSequenceNetwork){ context->enqueue(1, m_buffers, stream, nullptr); } 
else{ context->enqueue(batchSize, m_buffers, stream, nullptr); } 
hipStreamSynchronize(stream); hipStreamDestroy(stream); } 
MWTargetNetworkImpl::MWTargetNetworkImpl() : network(0) , builder(0) , 
engine(0) , context(0) , PiMNTwjpqwsGWomVWqdO(0) , m_buffers(0) { } void 
MWTargetNetworkImpl::preSetup() { PiMNTwjpqwsGWomVWqdO = new hipdnnHandle_t; 
hipdnnCreate(PiMNTwjpqwsGWomVWqdO); builder = createInferBuilder(gLogger); } 
void MWTargetNetworkImpl::allocate(int, int) { network = 
builder->createNetwork(); } void MWTargetNetworkImpl::postSetup(MWCNNLayer* 
layers[], int numLayers, int layerIdxs[], int portIdxs[], int numOuts) { 
markOutputs(layers, layerIdxs, numOuts); std::map<int, std::pair<float*, 
std::string> > buffers; setupBuffers(layers, layerIdxs, portIdxs, numOuts, buffers);
#if INT8_ENABLED
 bool useINT8 = builder->platformHasFastInt8(); if(!useINT8){ char buffer[100]; int numElem = sprintf(buffer,"#### INT8 mode is not supported on GPU available on the current machine! ####\n"); throw std::runtime_error(buffer); } else{ builder->setInt8Mode(1); } int trainBatchCount=0;  while(1) { char filename[500]; char filename1[500]; sprintf(filename,"|>targetdir<|/tensorrt/batch%d",trainBatchCount++); getValidDataPath(filename,filename1); FILE *fp = fopen(filename1,"rb"); if(fp==NULL) { trainBatchCount-=1; break; } fclose(fp); } BatchStream calibrationStream(trainBatchCount); Int8EntropyCalibrator calibrator(calibrationStream, 0); builder->setAverageFindIterations(1); builder->setMinFindIterations(1); builder->setDebugSync(true); builder->setInt8Calibrator(&calibrator);
#endif
#if FP16_ENABLED
 bool useFp16 = builder->platformHasFastFp16(); if(useFp16){ builder->setFp16Mode(1); } else{ printf("#### FP16 mode is not supported on GPU available on the current machine. Falling back to FP32 ####\n"); }
#endif
 builder->setMaxBatchSize(batchSize); unsigned int wsize = 1 << 30; 
builder->setMaxWorkspaceSize(wsize); engine = 
builder->buildCudaEngine(*network); hipError_t err = hipGetLastError(); if 
(err != hipSuccess) { builder->setMaxWorkspaceSize(1 << 26);  engine = 
builder->buildCudaEngine(*network);  hipError_t err = hipGetLastError(); if 
(err != hipSuccess) { builder = 0; engine = 0; network = 0; CUDA_CALL(err); } 
}  context = engine->createExecutionContext();  m_buffers = (void**) new 
float*[buffers.size()]; for (std::map<int, std::pair<float*, std::string> 
>::iterator it = buffers.begin(); it != buffers.end(); ++it) { int 
binding_index = engine->getBindingIndex((it->second.second).c_str()); 
m_buffers[binding_index] = it->second.first; } network->destroy(); } void 
MWTargetNetworkImpl::markOutputs(MWCNNLayer* layers[], int layerIdxs[], int 
numOuts){ for (int k = 0; k < numOuts; k++) { int layerIdx = layerIdxs[k]; 
MWCNNLayer* layer = layers[layerIdx]; ITensor* itensor = 
MWCNNLayerImpl::getITensor(layer->getOutputTensor(0)); char layerIdxStr[20]; 
sprintf(layerIdxStr, "output%d", layerIdx); itensor->setName(layerIdxStr); 
network->markOutput(*itensor); } } void 
MWTargetNetworkImpl::setupBuffers(MWCNNLayer* layers[], int layerIdxs[], int 
portIdxs[], int numOuts, std::map<int, std::pair<float*, std::string> > & 
buffers) { float* buffer = getBuffer(layers[0], 0, 0); auto inputITensor = 
MWCNNLayerImpl::getITensor(layers[0]->getOutputTensor(0)); buffers[0] = 
std::make_pair(buffer, std::string(inputITensor->getName())); for(int k = 0; k 
< numOuts; k++) { int layerIdx = layerIdxs[k]; MWCNNLayer* layer = 
layers[layerIdx]; ITensor* itensor = 
MWCNNLayerImpl::getITensor(layer->getOutputTensor(0)); float* buffer = 
getBuffer(layer, 0, portIdxs[k]); buffers[k+1] = std::make_pair(buffer, 
std::string(itensor->getName())); } } float* 
MWTargetNetworkImpl::getBuffer(MWCNNLayer* layer, int layerIdx, int portIdx) { 
MWTensor* opTensor = layer->getOutputTensor(portIdx); float* data = 
opTensor->getData<float>(); if (!data) { CUDA_CALL(hipMalloc((void**)&data, 
sizeof(float) * opTensor->getNumElements())); } opTensor->setData(data); return 
data; } hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
PiMNTwjpqwsGWomVWqdO; } void MWTargetNetworkImpl::deallocate() { if 
(m_buffers) { delete[] m_buffers; m_buffers = 0; } if (hipFree(0) != 
hipErrorDeinitialized) { if (context) { context->destroy(); context = 0; } 
if (engine) { engine->destroy(); engine = 0; } } } void 
MWTargetNetworkImpl::cleanup() { if (builder) { builder->destroy(); builder = 
0; } if (PiMNTwjpqwsGWomVWqdO) { hipdnnDestroy(*PiMNTwjpqwsGWomVWqdO); 
delete PiMNTwjpqwsGWomVWqdO; PiMNTwjpqwsGWomVWqdO = 0; } } 
MWTargetNetworkImpl::~MWTargetNetworkImpl() { }