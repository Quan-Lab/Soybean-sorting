//
//  _coder_myModelGPU_api.cu
//
//  Code generation for function '_coder_myModelGPU_api'
//


// Include files
#include "_coder_myModelGPU_api.h"
#include "myModelGPU.h"
#include "myModelGPU_data.h"
#include "rt_nonfinite.h"

// Function Declarations
static uint8_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId);
static uint8_T c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId);
static uint8_T emlrt_marshallIn(const mxArray *x, const char_T *identifier);
static const mxArray *emlrt_marshallOut(const real_T u);

// Function Definitions
static uint8_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId)
{
  uint8_T y;
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static uint8_T c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId)
{
  uint8_T ret;
  static const int32_T dims = 0;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "uint8", false, 0U,
    &dims);
  ret = *(uint8_T *)emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static uint8_T emlrt_marshallIn(const mxArray *x, const char_T *identifier)
{
  uint8_T y;
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(x), &thisId);
  emlrtDestroyArray(&x);
  return y;
}

static const mxArray *emlrt_marshallOut(const real_T u)
{
  const mxArray *y;
  const mxArray *m;
  y = NULL;
  m = emlrtCreateDoubleScalar(u);
  emlrtAssign(&y, m);
  return y;
}

void myModelGPU_api(const mxArray * const prhs[1], int32_T, const mxArray *plhs[1])
{
  uint8_T x;

  // Marshall function inputs
  x = emlrt_marshallIn(emlrtAliasP(prhs[0]), "x");

  // Invoke the target function
  // Marshall function outputs
  plhs[0] = emlrt_marshallOut(myModelGPU(x));
}

// End of code generation (_coder_myModelGPU_api.cu)
