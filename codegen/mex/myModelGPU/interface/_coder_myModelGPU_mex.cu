//
//  _coder_myModelGPU_mex.cu
//
//  Code generation for function '_coder_myModelGPU_mex'
//


// Include files
#include "_coder_myModelGPU_mex.h"
#include "_coder_myModelGPU_api.h"
#include "myModelGPU.h"
#include "myModelGPU_data.h"
#include "myModelGPU_initialize.h"
#include "myModelGPU_terminate.h"

// Function Declarations
MEXFUNCTION_LINKAGE void myModelGPU_mexFunction(int32_T nlhs, mxArray *plhs[1],
  int32_T nrhs, const mxArray *prhs[1]);

// Function Definitions
void myModelGPU_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T nrhs, const
  mxArray *prhs[1])
{
  const mxArray *outputs[1];

  // Check for proper number of arguments.
  if (nrhs != 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 1, 4, 7, "myModelGPU");
  }

  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 7,
                        "myModelGPU");
  }

  // Call the function.
  myModelGPU_api(prhs, nlhs, outputs);

  // Copy over outputs to the caller.
  emlrtReturnArrays(1, plhs, outputs);
}

void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs, const mxArray
                 *prhs[])
{
  mexAtExit(&myModelGPU_atexit);

  // Module initialization.
  myModelGPU_initialize();

  // Dispatch the entry-point.
  myModelGPU_mexFunction(nlhs, plhs, nrhs, prhs);

  // Module termination.
  myModelGPU_terminate();
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLS(&emlrtRootTLSGlobal, &emlrtContextGlobal, NULL, 1);
  return emlrtRootTLSGlobal;
}

// End of code generation (_coder_myModelGPU_mex.cu)
