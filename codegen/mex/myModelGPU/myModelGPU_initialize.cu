//
//  myModelGPU_initialize.cu
//
//  Code generation for function 'myModelGPU_initialize'
//


// Include files
#include "myModelGPU_initialize.h"
#include "_coder_myModelGPU_mex.h"
#include "myModelGPU.h"
#include "myModelGPU_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void myModelGPU_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (myModelGPU_initialize.cu)
