//
//  myModelGPU_terminate.cu
//
//  Code generation for function 'myModelGPU_terminate'
//


// Include files
#include "myModelGPU_terminate.h"
#include "_coder_myModelGPU_mex.h"
#include "myModelGPU.h"
#include "myModelGPU_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void myModelGPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void myModelGPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (myModelGPU_terminate.cu)
